#include "hip/hip_runtime.h"
#include <stdio.h>
#include<conio.h>
#include<math.h>
#include<hip/hip_runtime.h>

__global__ void max(__global float *A,int m,int incx)
{
int i=blockIdx.x*N+threadIdx.x;
int j=threadIdx.x;
__shared__ float asa[16];

asa[j]=0;
if(i*incx<m)
{
asa[j]=A[i*incx];
int k;
 if(j==0){int max=asa[0];
for(k=0;k<16;k++)
{
if(max<asa[k]){max=asa[k];	
}
}
A[get_group_id(0)]=max;
}
}
}

int main(int argc,char **argv)
{
 int n,incx,i,mm;
// printf("enter size of array");
//scanf("%d",&n);
n=atoi(argv[1]);
incx=atoi(argv[2]);
float *A = (float*)malloc(sizeof(float)*n);
int mm=n;
int x;
if(n%16==0){   
 x = n;
}
else{
x=(int)((floor(n/16)+1)*16);
}
int xx=x/16;
int *dev_a;
// allocate the memory on the GPU
HANDLE_ERROR( hipMalloc( (void**)&dev_a,n * sizeof(float) ) );

for(i = 0; i < n; i++) 
{
A[i] =i;
}


//copy the arrays
HANDLE_ERROR(hipMemcpy(dev_a,a,n*sizeof(float),hipMemcpyHostToDevice));

while(mm!=0)
{
max<<<x,16>>> (n,dev_a,incx);

if(x%16!=0)
{
x=(int)(floor(xx/16)+1)*16;
}
else
{
x=xx;
}
xx=x/16;
mm=mm/16;
incx=1;

}
HANDLE_ERROR(hipMemcpy(a,dev_a,a*sizeof(float),hipMemcpyDeviceToHost));
 
printf("max=\t");  
  
printf("%f\t",A[0]);

hipFree(dev_a);


return 0;
}
}
