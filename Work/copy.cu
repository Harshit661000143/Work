#include "hip/hip_runtime.h"
#include <stdio.h>
#include<conio.h>
#include<math.h>
#include<hip/hip_runtime.h>
__global__ void copy((int n,__global float *A,int incx, __global float *B,int incy)
 
{
int tx=blockIdx.x*N+threadIdx.x;
if((tx*incx<n)&&(tx*incy<n))
{A[tx*incy]= B[tx*incx];
}
}

int main( int argc,char **argv ) {
 int n,i,incx,incy;
float alpha;
   n=atoi(argv[1]);
incx=atoi(argv[2]);
incy=atoi(argv[3]);
  float *A = (float*)malloc(sizeof(float)*n);
    float *B = (float*)malloc(sizeof(float)*n);    
    for(i = 0; i < n; i++) 
{
        A[i] = 0.0;
        B[i] = 1.0;
    }

int x;
if(n%16==0){   
 x = n;

}
else{
x=(int)((floor(n/16)+1)*16);
}

int *dev_a, *dev_b, *dev_c;
// allocate the memory on the GPU
HANDLE_ERROR( hipMalloc( (void**)&dev_a,n * sizeof(float) ) );
HANDLE_ERROR( hipMalloc( (void**)&dev_b,n * sizeof(float) ) );

// fill the arrays 'a' and 'b' on the CPU
    for(i = 0; i < n; i++) 
{
        A[i] = 0.0;
        B[i] = 1.0;
    }

//copy the arrays
HANDLE_ERROR(hipMemcpy(dev_a,a,n*sizeof(float),hipMemcpyHostToDevice));
HANDLE_ERROR(hipMemcpy(dev_b,b,n*sizeof(float),hipMemcpyHostToDevice));
copy<<<x,16>>>(n,dev_a,incx,dev_b,incy);
HANDLE_ERROR(hipMemcpy(a,dev_a,n*sizeof(float),hipMemcpyDeviceToHost));
 
printf("A=\t");  
  for(i = 0; i < n; i++)
{
printf("%f\t",A[i]);
}

printf("\nB=\t");  
  for(i = 0; i <n ; i++)
{
printf("%f\t",B[i]);
}
hipFree(dev_a);
hipFree(dev_b);

return 0;
}
}
